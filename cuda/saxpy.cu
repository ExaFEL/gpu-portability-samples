
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void saxpy(const size_t num_elements, const float alpha,
                      const float *x, const float *y, float *z) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (idx < num_elements) z[idx] += alpha * x[idx] + y[idx];
}

int main() {
  size_t num_elements = 1 << 20;
  size_t buffer_size = num_elements * sizeof(float);

  float *x = (float *)malloc(buffer_size);
  float *y = (float *)malloc(buffer_size);
  float *z = (float *)malloc(buffer_size);

  float *d_x, *d_y, *d_z;
  hipMalloc(&d_x, buffer_size);
  hipMalloc(&d_y, buffer_size);
  hipMalloc(&d_z, buffer_size);

  for (size_t idx = 0; idx < num_elements; idx++) {
    x[idx] = 1.0f;
    y[idx] = 2.0f;
    z[idx] = 0.0f;
  }

  hipMemcpy(d_x, x, buffer_size, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, buffer_size, hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, buffer_size, hipMemcpyHostToDevice);

  saxpy<<<(num_elements + 255) / 256, 256>>>(num_elements, 2.0f, d_x, d_y, d_z);

  hipMemcpy(z, d_z, buffer_size, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  float error = 0.0;
  for (size_t idx = 0; idx < num_elements; idx++) {
    error = fmax(error, fabs(z[idx] - 4.0f));
  }
  printf("error: %e\n", error);

  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);

  free(x);
  free(y);
  free(z);

  return 0;
}
